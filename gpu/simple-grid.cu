
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define dwellLimit 0x0000000400000000
#define C_EQUIVALENCE_THRESHOLD pow(2, -16)
#define O_EQUIVALENCE_THRESHOLD pow(2, -32)
#define S_SEED 0xa6ceb87d4389e4ef
#define FILE_OUTPUT 1

#if __unix__
#define RNGSEED 1
#else
#define RNGSEED 0
#endif

enum pointState { MEMBER,
                  NOT_A_MEMBER,
                  UNDECIDED,
                  OUT_OF_RANGE,
                  PUSH_TO_CPU };
char membership(double re, double im);

#define GRID_SIZE 262144
#define PARALLEL_SIZE 100
#define POINTS_PER_CUDA_THREAD 1000

// from v8: https://github.com/v8/v8/blob/main/src/base/utils/random-number-generator.h#L119
uint64_t state0 = 1;
uint64_t state1 = 2;
uint64_t xorshift128plus() {
    uint64_t s1 = state0;
    uint64_t s0 = state1;
    state0 = s0;
    s1 ^= s1 << 23;
    s1 ^= s1 >> 17;
    s1 ^= s0;
    s1 ^= s0 >> 26;
    state1 = s1;
    return state0 + state1;
}

void xorshift128plusCUDA(uint64_t* state0, uint64_t* state1, uint64_t* result) {
    uint64_t s1 = *state0;
    uint64_t s0 = *state1;
    *state0 = s0;
    s1 ^= s1 << 23;
    s1 ^= s1 >> 17;
    s1 ^= s0;
    s1 ^= s0 >> 26;
    *state1 = s1;
    *result = *state0 + *state1;
}

void prnginit() {
    for (int i = 0; i < 128; i++)
        xorshift128plus();
}

// mask to [0,1)
inline double _01() {
    uint64_t u64 = 0x3FF0000000000000ULL | (xorshift128plus() >> 12);
    return *(double*)&u64 - 1.0;
}

void reseed() {
    uint64_t urs0, urs1;

    if (RNGSEED) {
        FILE* randsource;
        randsource = fopen("/dev/urandom", "r");
        fread(&urs0, 1, 8, randsource);
        fread(&urs1, 1, 8, randsource);
        fclose(randsource);
    }

    state0 ^= time(NULL) ^ urs0;
    state1 ^= time(NULL) ^ S_SEED ^ urs1;
}

void inspectPRNGstate() {
    printf("PRNG SEED 0: %llx\n", state0);
    printf("PRNG SEED 1: %llx\n", state1);
}

// extract the first bit of a double
char fneg(double x) {
    return (char)((*(uint64_t*)&x) >> 63);
}

inline char membershipt(double re, double im, uint64_t numiters) {
    if (im < -1.15 || im > 1.15 || re < -2.0 || re > 0.49 || re * re + im * im > 4.0) {
        return NOT_A_MEMBER;
    }

    // https://www.desmos.com/calculator/rmldovq5x5
    // testing main cardioid and main bulb
    if (im < 0.25 && re < -0.75 && re > -1.25) {
        const double xp1 = re + 1;
        if (fneg(xp1 * xp1 + im * im - 0.0625)) {
            return MEMBER;
        }
    } else if (re > -0.75 && im < 0.65 && re < 0.375) {
        const double adjx = re - 0.25;
        const double adjx2py2 = adjx * adjx + im * im;
        const double firstterm = adjx2py2 + 2 * 0.25 * adjx;
        if (fneg(firstterm * firstterm - 0.25 * adjx2py2)) {
            return MEMBER;
        }
    }

    const double cRe = re, cIm = im;
    double pRe, pIm, pobRe;
    double obRe = re, obIm = im;

    for (uint64_t i = 0; i < numiters; i++) {
        pRe = re;
        pIm = im;
        re = re * re - im * im + cRe;
        im = 2.0 * pRe * im + cIm;
        if (i % 5 == 1) {
            if (re * re + im * im > 4.0) return NOT_A_MEMBER;
            if (fabs(pRe - re) < C_EQUIVALENCE_THRESHOLD && fabs(pIm - im) < C_EQUIVALENCE_THRESHOLD) return MEMBER;
        }
        if (i % 2) {
            pobRe = obRe;
            obRe = obRe * obRe - obIm * obIm + cRe;
            obIm = 2 * pobRe * obIm + cIm;
            if (fabs(obRe - re) < O_EQUIVALENCE_THRESHOLD && fabs(obIm - im) < O_EQUIVALENCE_THRESHOLD) return MEMBER;
        }
    }

    return UNDECIDED;
}

__global__ void membershipKernel(uint64_t startPos, uint32_t threadLoopLength, char* resultPtr) {
    int threadId = 0; // TODO thread index
    for(int i = 0; i < threadLoopLength; i++) {

    }
}

void cudamanagement(uint64_t startPos, uint32_t threadLoopLength, uint32_t parallelSize, char* resultPtr) {
    // TODO call kernel here
}

int main(int argc, char** argv) {
    reseed();
    inspectPRNGstate();
    prnginit();
    inspectPRNGstate();

    unsigned long gridTested = 0;
    const double deltaRe = 2.49 / GRID_SIZE;
    const double deltaIm = 1.15 / GRID_SIZE;

    const uint64_t totalPointCount = GRID_SIZE * GRID_SIZE;
    uint64_t prngseed0[PARALLEL_SIZE];
    uint64_t prngseed1[PARALLEL_SIZE];
    char result[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD];
    double cpuQueueRe[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD];
    double cpuQueueIm[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD];

    while (1) {
        unsigned long startTime = time(NULL);  // timestamps will not be accurate, timer itself will be close enough
        uint64_t member = 0,
                 notmem = 0,
                 undeci = 0,
                 tested = 0;

        for (uint64_t position = 0; position < totalPointCount; position += PARALLEL_SIZE * POINTS_PER_CUDA_THREAD) {
            reseed();
            for (uint32_t i = 0; i < PARALLEL_SIZE; i++) {
                prngseed0[i] = xorshift128plus();
                prngseed1[i] = xorshift128plus();
            }

            cudamanagement(position, POINTS_PER_CUDA_THREAD, PARALLEL_SIZE, &result[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD]);

            for(uint32_t i = 0; i < PARALLEL_SIZE; i++) {
                char wv = result[i];
                member += wv == MEMBER;
                notmem += wv == NOT_A_MEMBER;
            }
        }

        if (FILE_OUTPUT == 0) continue;
        FILE* fp;
        fp = fopen("log.txt", "a");
        fprintf(fp, "%lu, %lu, %llu, %llu, %llu\n", (unsigned long)time(NULL), startTime, member, notmem, undeci);
        fclose(fp);

        gridTested++;

        printf("GRID DONE %llu points tested; grid number %lu\n", tested, gridTested);
        printf("times:               %llu %llu\n", startTime, time(NULL));
        printf("estimated area H:    %lf\n", (double)(member + undeci) / tested * 5.727);
        printf("estimated area L:    %lf\n", (double)(member) / tested * 5.727);
        printf("\n");
    }
    
    return 0;
}
