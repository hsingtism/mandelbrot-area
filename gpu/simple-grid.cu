
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define dwellLimit 0x0000000400000000
#define C_EQUIVALENCE_THRESHOLD pow(2, -16)
#define O_EQUIVALENCE_THRESHOLD pow(2, -32)
#define S_SEED 0xa6ceb87d4389e4ef
#define FILE_OUTPUT 1

#if __unix__
#define RNGSEED 1
#else
#define RNGSEED 0
#endif

enum pointState { reservedForError, //errors will tend to have zeros
                  MEMBER,           //only defined for function return, be careful to context
                  NOT_A_MEMBER,
                  UNDECIDED,
                  OUT_OF_RANGE };

#define GRID_SIZE 8192
#define CUDA_GRID_COUNT 128
#define CUDA_THREAD_COUNT 512
#define POINTS_PER_CUDA_THREAD 512
#define DEVICE_ITERS 512

#define PARALLEL_SIZE CUDA_THREAD_COUNT * CUDA_GRID_COUNT
__managed__ double deltaRe = 2.49 / GRID_SIZE;
__managed__ double deltaIm = 1.15 / GRID_SIZE;

// from v8: https://github.com/v8/v8/blob/main/src/base/utils/random-number-generator.h#L119
uint64_t state0 = 1;
uint64_t state1 = 2;
uint64_t xorshift128plus() {
    uint64_t s1 = state0;
    uint64_t s0 = state1;
    state0 = s0;
    s1 ^= s1 << 23;
    s1 ^= s1 >> 17;
    s1 ^= s0;
    s1 ^= s0 >> 26;
    state1 = s1;
    return state0 + state1;
}

__device__ void xorshift128plusCUDA(uint64_t* state0, uint64_t* state1, uint64_t* result) {
    uint64_t s1 = *state0;
    uint64_t s0 = *state1;
    *state0 = s0;
    s1 ^= s1 << 23;
    s1 ^= s1 >> 17;
    s1 ^= s0;
    s1 ^= s0 >> 26;
    *state1 = s1;
    *result = *state0 + *state1;
}

void prnginit() {
    for (int i = 0; i < 128; i++)
        xorshift128plus();
}

// mask to [0,1)
inline double _01() {
    uint64_t u64 = 0x3FF0000000000000ULL | (xorshift128plus() >> 12);
    return *(double*)&u64 - 1.0;
}

__device__ double _01MaskOnly(uint64_t src) {
    uint64_t u64 = 0x3FF0000000000000ULL | (src >> 12);
    return *(double*)&u64 - 1.0;
}

void reseed() {
    uint64_t urs0, urs1;

    if (RNGSEED) {
        FILE* randsource;
        randsource = fopen("/dev/urandom", "r");
        fread(&urs0, 1, 8, randsource);
        fread(&urs1, 1, 8, randsource);
        fclose(randsource);
    }

    state0 ^= time(NULL) ^ urs0;
    state1 ^= time(NULL) ^ S_SEED ^ urs1;
}

void inspectPRNGstate() {
    printf("PRNG SEED 0: %llx\n", state0);
    printf("PRNG SEED 1: %llx\n", state1);
}

// extract the first bit of a double
__device__ __host__ char fneg(double x) {
    return (char)((*(uint64_t*)&x) >> 63);
}

__device__ __host__ char membershipt(double re, double im, uint64_t numiters) {
    if (im < -1.15 || im > 1.15 || re < -2.0 || re > 0.49 || re * re + im * im > 4.0) {
        return NOT_A_MEMBER;
    }

    // https://www.desmos.com/calculator/rmldovq5x5
    // testing main cardioid and main bulb
    if (im < 0.25 && re < -0.75 && re > -1.25) {
        const double xp1 = re + 1;
        if (fneg(xp1 * xp1 + im * im - 0.0625)) {
            return MEMBER;
        }
    } else if (re > -0.75 && im < 0.65 && re < 0.375) {
        const double adjx = re - 0.25;
        const double adjx2py2 = adjx * adjx + im * im;
        const double firstterm = adjx2py2 + 2 * 0.25 * adjx;
        if (fneg(firstterm * firstterm - 0.25 * adjx2py2)) {
            return MEMBER;
        }
    }

    const double cRe = re, cIm = im;
    double pRe, pIm, pobRe;
    double obRe = re, obIm = im;

    for (uint64_t i = 0; i < numiters; i++) {
        pRe = re;
        pIm = im;
        re = re * re - im * im + cRe;
        im = 2.0 * pRe * im + cIm;
        if (i % 5 == 1) {
            if (re * re + im * im > 4.0) return NOT_A_MEMBER;
            if (fabs(pRe - re) < C_EQUIVALENCE_THRESHOLD && fabs(pIm - im) < C_EQUIVALENCE_THRESHOLD) return MEMBER;
        }
        if (i % 2) {
            pobRe = obRe;
            obRe = obRe * obRe - obIm * obIm + cRe;
            obIm = 2 * pobRe * obIm + cIm;
            if (fabs(obRe - re) < O_EQUIVALENCE_THRESHOLD && fabs(obIm - im) < O_EQUIVALENCE_THRESHOLD) return MEMBER;
        }
    }

    return UNDECIDED;
}

__global__ void membershipKernel(uint64_t startPos, uint32_t threadLoopLength, char* resultPtr) {
    const int threadId = blockIdx.x * CUDA_GRID_COUNT + threadIdx.x; 
    const int resultIndexStart = threadId * POINTS_PER_CUDA_THREAD;

    uint64_t prngState0 = startPos + resultIndexStart;
    uint64_t prngState1 = (startPos + resultIndexStart) << 32;
    uint64_t prngresRe = 0;
    uint64_t prngresIm = 0;

    for(int i = 0; i < threadLoopLength; i++) {
        if(startPos + threadLoopLength > GRID_SIZE * GRID_SIZE) {
            resultPtr[resultIndexStart + i] = OUT_OF_RANGE;
            continue;
        }
        xorshift128plusCUDA(&prngState0, &prngState1, &prngresRe);
        xorshift128plusCUDA(&prngState0, &prngState1, &prngresIm);
        resultPtr[resultIndexStart + i] = membershipt(
            ((startPos + resultIndexStart + i) % GRID_SIZE) + _01MaskOnly(prngresRe) * deltaRe,
            ((startPos + resultIndexStart + i) / GRID_SIZE) + _01MaskOnly(prngresIm) * deltaIm,
            DEVICE_ITERS);
    }
}

void cudamanagement(uint64_t startPos, uint32_t threadLoopLength, char* resultPtr) {
    membershipKernel<<<CUDA_GRID_COUNT,CUDA_THREAD_COUNT>>>(
        startPos, threadLoopLength, resultPtr
    );
}

int main(int argc, char** argv) {
    reseed();
    inspectPRNGstate();
    prnginit();
    inspectPRNGstate();

    unsigned long gridTested = 0;

    const uint64_t totalPointCount = GRID_SIZE * GRID_SIZE;

    char* result;
    char* result_d;

    result = (char *)malloc(PARALLEL_SIZE * POINTS_PER_CUDA_THREAD);
    hipMalloc((void **)&result_d, PARALLEL_SIZE * POINTS_PER_CUDA_THREAD);

    double cpuQueueRe[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD];
    double cpuQueueIm[PARALLEL_SIZE * POINTS_PER_CUDA_THREAD];

    while (1) {
        unsigned long startTime = time(NULL);  // timestamps will not be accurate, timer itself will be close enough
        uint64_t member = 0,
                 notmem = 0,
                 undeci = 0,
                 tested = 0;

        for (uint64_t position = 0; position < totalPointCount; position += PARALLEL_SIZE * POINTS_PER_CUDA_THREAD) {
            cudamanagement(position, POINTS_PER_CUDA_THREAD, result_d);
            hipMemcpy(result, result_d, PARALLEL_SIZE * POINTS_PER_CUDA_THREAD, hipMemcpyDeviceToHost);

            uint32_t CPUArrayPosition = 0;

            for(uint32_t i = 0; i < PARALLEL_SIZE; i++) {
                char wv = result[i];
                if(wv == NOT_A_MEMBER) { // no branchless here since some conditions needs to be executed
                    notmem++;
                } else if (wv == MEMBER) {
                    member++;
                } else if (wv == UNDECIDED) {
                    if(i == 0) reseed();
                    cpuQueueIm[CPUArrayPosition] = (position + i) / GRID_SIZE * deltaIm * _01();
                    cpuQueueRe[CPUArrayPosition] = (position + i) % GRID_SIZE * deltaRe * _01();
                    CPUArrayPosition++;
                } else if (wv == OUT_OF_RANGE) {
                    continue;
                } else {
                    printf("unexpected cuda return");
                }
            }

            while(CPUArrayPosition--) {
                char memdat = membershipt(
                    cpuQueueRe[CPUArrayPosition],
                    cpuQueueIm[CPUArrayPosition],
                    dwellLimit
                );
                member += memdat == MEMBER;
                notmem += memdat == NOT_A_MEMBER;
                undeci += memdat == UNDECIDED;
            }
        }

        if (FILE_OUTPUT == 0) continue;
        FILE* fp;
        fp = fopen("log.txt", "a");
        fprintf(fp, "%lu, %lu, %llu, %llu, %llu\n", (unsigned long)time(NULL), startTime, member, notmem, undeci);
        fclose(fp);

        gridTested++;

        printf("GRID DONE %llu points tested; grid number %lu\n", tested, gridTested);
        printf("times:               %llu %llu\n", startTime, time(NULL));
        printf("estimated area H:    %lf\n", (double)(member + undeci) / tested * 5.727);
        printf("estimated area L:    %lf\n", (double)(member) / tested * 5.727);
        printf("\n");
    }
    
    return 0;
}
